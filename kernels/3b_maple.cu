#include "hip/hip_runtime.h"
// #define DEBUG

__global__ void evaluate_3b_maple(
    double * a, double * x, double * g, double * energy_buffer
)
{

#include "3b_maple_polynomial.cu"

energy_buffer[0] = energy;
}

void launch_evaluate_3b_maple(double * a, double * x, double * g, double * e) {
    evaluate_3b_maple<<<1,400>>>(a, x, g, e);
    hipDeviceSynchronize();
}
